#include "hip/hip_runtime.h"
#include "shared_header.h"

__global__ void soa_update(float *posx, float *posy, float *posz, float *velx, float *vely, float *velz, float *mass) {
    // one kernel per particle
    // 1024 threads per particle
    // Todo use constant mem for save original points
    // Note 32 is size of warp
    int id = blockIdx.x;
    int other = threadIdx.x;
    __shared__ float x_values[1024];
    __shared__ float y_values[1024];
    __shared__ float z_values[1024];
    x_values[other] = 0;
    y_values[other] = 0;
    z_values[other] = 0;
    float posix = posx[id];
    float posiy = posy[id];
    float posiz = posz[id];

    for (std::size_t j = other; j < PROBLEMSIZE; j += 1024) {
        const float xdistance = posix - posx[j];
        const float ydistance = posiy - posy[j];
        const float zdistance = posiz - posz[j];
        const float xdistanceSqr = xdistance * xdistance;
        const float ydistanceSqr = ydistance * ydistance;
        const float zdistanceSqr = zdistance * zdistance;
        const float distSqr = EPS2 + xdistanceSqr + ydistanceSqr + zdistanceSqr;
        const float distSixth = distSqr * distSqr * distSqr;
        const float invDistCube = 1.0f / std::sqrt(distSixth);
        const float sts = mass[j] * invDistCube * TIMESTEP;
        x_values[other] += xdistanceSqr * sts;
        y_values[other] += ydistanceSqr * sts;
        z_values[other] += zdistanceSqr * sts;
    }
    // reduce to one
    // Todo improve with half steps
    SYNC_THREADS;
    if (id == 0) {
        for (int j = 1; j < 1024; j++) {

            x_values[0] += x_values[j];
            y_values[0] += y_values[j];
            z_values[0] += z_values[j];
        }
        velx[id] += x_values[0];
        vely[id] += y_values[0];
        velz[id] += z_values[0];
    }
    SYNC_THREADS;
}

__global__ void soa_move(float *posx, float *posy, float *posz, float *velx,
                         float *vely, float *velz) {
    int id = LINEAR_ID;
    if ( id < kProblemSize) {
        posx[id] += velx[id] * TIMESTEP;
        posy[id] += vely[id] * TIMESTEP;
        posz[id] += velz[id] * TIMESTEP;
    }
}

void soa_run() {

    float *posx_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *posy_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *posz_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *velx_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *vely_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *velz_h = (float *) malloc(sizeof(float) * kProblemSize);
    float *mass_h = (float *) malloc(sizeof(float) * kProblemSize);
    srand(NULL);
    for (std::size_t i = 0; i < kProblemSize; ++i) {
        posx_h[i] = (float) rand();
        posy_h[i] = (float) rand();
        posz_h[i] = (float) rand();
        velx_h[i] = (float) rand() / 10.0f;
        vely_h[i] = (float) rand() / 10.0f;
        velz_h[i] = (float) rand() / 10.0f;
        mass_h[i] = (float) rand() / 100.0f;
    }

    float *posx_d;
    float *posy_d;
    float *posz_d;
    float *velx_d;
    float *vely_d;
    float *velz_d;
    float *mass_d;

    HANDLE_ERROR(hipMalloc(&posx_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&posy_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&posz_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&velx_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&vely_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&velz_d, kProblemSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&mass_d, kProblemSize * sizeof(float)));
    // copy points to Device
    HANDLE_ERROR(hipMemcpy(posx_d, posx_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(posy_d, posy_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(posz_d, posz_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(velx_d, velx_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vely_d, vely_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(velz_d, velz_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mass_d, mass_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice));

    //
    hipEvent_t start, end;
    hipEvent_t start2, end2;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventCreate(&start2));
    HANDLE_ERROR(hipEventCreate(&end2));

    for (std::size_t s = 0; s < STEPS; ++s) {
        HANDLE_ERROR(hipEventRecord(start, 0));

        soa_update<<<kProblemSize, 1024>>>(posx_d, posy_d, posz_d, velx_d, vely_d, velz_d, mass_d);
        hipEventRecord(end, 0);

        hipEventRecord(start2, 0);
        soa_move<<<(kProblemSize+1023) / 1024, 1024>>>(posx_d, posy_d, posz_d, velx_d, vely_d, velz_d);
        hipEventRecord(end2, 0);
        HANDLE_ERROR(hipEventSynchronize(end2));
        // HANDLE_LAST_ERROR;
        float time;
        hipEventElapsedTime(&time, start, end);
        float time2;
        hipEventElapsedTime(&time2, start2, end2);
        std::cout << "SoA\t" << time / 1 << "ms" << '\t' << time2 / 1 << "ms"
                  << '\n';
    }
    hipMemcpy(posx_h, posx_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(posy_h, posy_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(posz_h, posz_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(velx_h, velx_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vely_h, vely_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(velz_h, velz_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mass_h, mass_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
}