#include "hip/hip_runtime.h"
#include "shared_header.h"


__global__ void soa_update(float* posx, float* posy, float* posz, float* velx, float* vely, float* velz, float* mass) {
    // one kernel per particle
    // 1024 threads per particle
    // Todo use constant mem for save original points
    // Note 32 is size of warp
    int id = blockIdx.x;
    int other = threadIdx.x;
    __shared__ float x_values[1024];
    __shared__ float y_values[1024];
    __shared__ float z_values[1024];
    x_values[id] = 0;
    y_values[id] = 0;
    z_values[id] = 0;


    for (std::size_t j = other; j < PROBLEMSIZE; j += 1024) {
        const float xdistance = posx[id] - posx[j];
        const float ydistance = posy[id] - posy[j];
        const float zdistance = posz[id] - posz[j];
        const float xdistanceSqr = xdistance * xdistance;
        const float ydistanceSqr = ydistance * ydistance;
        const float zdistanceSqr = zdistance * zdistance;
        const float distSqr = EPS2 + xdistanceSqr + ydistanceSqr + zdistanceSqr;
        const float distSixth = distSqr * distSqr * distSqr;
        const float invDistCube = 1.0f / std::sqrt(distSixth);
        const float sts = mass[other] * invDistCube * TIMESTEP;
        x_values[id] += xdistanceSqr * sts;
        y_values[id] += xdistanceSqr * sts;
        z_values[id] += xdistanceSqr * sts;
    }
    // reduce to one
    SYNC_THREADS;
    if (id == 0) {
      printf("sum");
        for (int j = 1; j < 1024; j++) {

            x_values[0] += x_values[j];
            y_values[0] += y_values[j];
            z_values[0] += z_values[j];
        }
        velx[id] += x_values[0];
        vely[id] += y_values[0];
        velz[id] += z_values[0];
    }
    SYNC_THREADS;
}
/*
__global__ void move() {
  int id = LINEAR_ID;
  for (std::size_t i = id; i < kProblemSize; i += 1024) {
    posx[i] += velx[i] * kTimestep;
    posy[i] += vely[i] * kTimestep;
    posz[i] += velz[i] * kTimestep;
  }
}*/

void soa_run() {

    float* posx_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* posy_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* posz_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* velx_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* vely_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* velz_h = (float*)malloc(sizeof(float) * kProblemSize);
    float* mass_h = (float*)malloc(sizeof(float) * kProblemSize);
    srand(NULL);
    for (std::size_t i = 0; i < kProblemSize; ++i) {
        posx_h[i] = (float)rand();
        posy_h[i] = (float)rand();
        posz_h[i] = (float)rand();
        velx_h[i] = (float)rand() / 10.0f;
        vely_h[i] = (float)rand() / 10.0f;
        velz_h[i] = (float)rand() / 10.0f;
        mass_h[i] = (float)rand() / 100.0f;
    }

    float* posx_d;
    float* posy_d;
    float* posz_d;
    float* velx_d;
    float* vely_d;
    float* velz_d;
    float* mass_d;

    hipMalloc(&posx_d, kProblemSize * sizeof(float));
    hipMalloc(&posy_d, kProblemSize * sizeof(float));
    hipMalloc(&posz_d, kProblemSize * sizeof(float));
    hipMalloc(&velx_d, kProblemSize * sizeof(float));
    hipMalloc(&vely_d, kProblemSize * sizeof(float));
    hipMalloc(&velz_d, kProblemSize * sizeof(float));
    hipMalloc(&mass_d, kProblemSize * sizeof(float));
    // copy points to Device
    hipMemcpy(posx_d, posx_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(posy_d, posy_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(posz_d, posz_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(velx_d, velx_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vely_d, vely_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(velz_d, velz_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mass_d, mass_h, kProblemSize * sizeof(float), hipMemcpyHostToDevice);

    //
    hipEvent_t start, end;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));

    // for (std::size_t s = 0; s <  kSteps; ++s) {
    hipEventRecord(start, 0);
    std::cout << "Kernel\n";
    soa_update << <kProblemSize, 1024 , sizeof (float)*3*kProblemSize>> > (posx_d, posy_d, posz_d, velx_d, vely_d, velz_d, mass_d);
    hipEventRecord(end, 0);

    hipEventSynchronize(end);
    HANDLE_LAST_ERROR;
    float time;
    hipEventElapsedTime(&time, start, end);

  hipMemcpy(posx_h, posx_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(posy_h, posy_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(posz_h, posz_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(velx_h, velx_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vely_h, vely_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(velz_h, velz_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(mass_h, mass_d, kProblemSize * sizeof(float), hipMemcpyDeviceToHost);

    // move<<<kProblemSize / 1024, 1024>>>(posx.data(), posy.data(), posz.data(),
    //                                    velx.data(), vely.data(), velz.data());
    // sumMove += watch.elapsedAndReset();
    // }
    std::cout << "SoA\t" << time / kSteps << "ms" << '\t' << '\n';
}