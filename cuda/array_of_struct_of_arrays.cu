#include "hip/hip_runtime.h"
#include "shared_header.h"

struct particle_block        // deleted alignas(64)
{
    struct
    {
        float x[LANES];
        float y[LANES];
        float z[LANES];
    } pos;
    struct
    {
        float x[LANES];
        float y[LANES];
        float z[LANES];
    } vel;
    float mass[LANES];
};

__device__ inline void pPInteraction(
        float piposx,
        float piposy,
        float piposz,
        float* pivelx,
        float* pively,
        float* pivelz,
        float pjposx,
        float pjposy,
        float pjposz,
        float pjmass)
        {
    float xdistance = piposx - pjposx;
    float ydistance = piposy - pjposy;
    float zdistance = piposz - pjposz;
    xdistance *= xdistance;
    ydistance *= ydistance;
    zdistance *= zdistance;
    const float distSqr = EPS2 + xdistance + ydistance + zdistance;
    const float distSixth = distSqr * distSqr * distSqr;
    const float invDistCube = 1.0f / std::sqrt(distSixth);
    const float sts = pjmass * invDistCube * TIMESTEP;
    *pivelx += xdistance * sts;
    *pively += ydistance * sts;
    *pivelz += zdistance * sts;
}

__global__ void aosoa_update(particle_block* particles){
    particle_block mainBlock = particles[blockIdx.x];
    int mainLane = threadIdx.x;
    for (int otherBlockIndex = 0; otherBlockIndex < BLOCKS; ++otherBlockIndex){
        for (int otherLane = 0; otherLane < LANES; ++otherLane){
            particle_block otherBlock = particles[otherBlockIndex];
        pPInteraction( mainBlock.pos.x[mainLane],
            mainBlock.pos.y[mainLane],
            mainBlock.pos.z[mainLane],
            &mainBlock.vel.x[mainLane],
            &mainBlock.vel.y[mainLane],
            &mainBlock.vel.z[mainLane],
            otherBlock.pos.x[otherLane],
            otherBlock.pos.y[otherLane],
            otherBlock.pos.z[otherLane],
            otherBlock.mass[otherLane]);
        }
    }
}


__global__ void aosoa_move(struct particle_block* particle_block){
    int bi = blockIdx.x;
    int i = threadIdx.x;

    struct particle_block block = particle_block[bi];
    block.pos.x[i] += block.vel.x[i] * TIMESTEP;
    block.pos.y[i] += block.vel.y[i] * TIMESTEP;
    block.pos.z[i] += block.vel.z[i] * TIMESTEP;
}



void aosoa_run(){

    // init event management
    hipEvent_t start_update, stop_update;
    hipEventCreate(&start_update);
    hipEventCreate(&stop_update);

    hipEvent_t start_move, stop_move;
    hipEventCreate(&start_move);
    hipEventCreate(&stop_move);

    // "allocate" mem
    struct particle_block particle_block_host[BLOCKS];
    struct particle_block* particle_block_device;

    // fill with random values
    // iterate over the structs 'stru' in the array
    for (int stru = 0; stru < ( sizeof(particle_block_host) / sizeof(struct particle_block) ); ++stru) {
        // iterate over pos-array inside the struct, then over the vel-arrays, then mass-array
        for (int l = 0; l < LANES; ++l) {
            particle_block_host[stru].pos.x[l] = (float)rand();
            particle_block_host[stru].pos.y[l] = (float)rand();
            particle_block_host[stru].pos.z[l] = (float)rand();

            particle_block_host[stru].vel.x[l] = (float)rand() / 10.0f;
            particle_block_host[stru].vel.y[l] = (float)rand() / 10.0f;
            particle_block_host[stru].vel.z[l] = (float)rand() / 10.0f;

            particle_block_host[stru].mass[l] = (float)rand() / 100.0f;
        }
    }

    int datasize = sizeof(particle_block_host);
    HANDLE_ERROR( hipMalloc(&particle_block_device, datasize) );
    HANDLE_ERROR( hipMemcpy(particle_block_device, particle_block_host, datasize, hipMemcpyHostToDevice) );


    float time_update, time_move;
    for (int i=0; i< STEPS; ++i) {
        // call update
        hipEventRecord(start_update, 0);
        aosoa_update<<<(PROBLEMSIZE +LANES-1)/LANES, LANES>>>(particle_block_device);
        HANDLE_LAST_ERROR;
        hipEventRecord(stop_update, 0);
        // call move
        hipEventRecord(start_move, 0);
        aosoa_move<<<(PROBLEMSIZE +LANES-1)/LANES, LANES>>>(particle_block_device);
        HANDLE_LAST_ERROR;
        hipEventRecord(stop_move, 0);

        hipEventSynchronize(stop_move);
        hipEventElapsedTime(&time_update, start_update, stop_update);
        printf("AoSoA\t%fms\t%fms\n", time_update, time_move);
        hipEventElapsedTime(&time_move, start_move, stop_move);
    }

    // maybe write back

    // free mem
    HANDLE_ERROR( hipFree(particle_block_device) );
    HANDLE_ERROR( hipDeviceReset() );
}
