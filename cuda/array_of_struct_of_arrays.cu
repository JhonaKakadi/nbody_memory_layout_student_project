#include "hip/hip_runtime.h"
#include "shared_header.h"

struct particle_block        // deleted alignas(64)
{
    struct {
        float x[LANES];
        float y[LANES];
        float z[LANES];
    } pos;
    struct {
        float x[LANES];
        float y[LANES];
        float z[LANES];
    } vel;
    float mass[LANES];
};

__device__ inline void pPInteraction(
        float piposx,
        float piposy,
        float piposz,
        float *pivelx,
        float *pively,
        float *pivelz,
        float pjposx,
        float pjposy,
        float pjposz,
        float pjmass) {
    float xdistance = piposx - pjposx;
    float ydistance = piposy - pjposy;
    float zdistance = piposz - pjposz;
    xdistance *= xdistance;
    ydistance *= ydistance;
    zdistance *= zdistance;
    const float distSqr = EPS2 + xdistance + ydistance + zdistance;
    const float distSixth = distSqr * distSqr * distSqr;
    const float invDistCube = 1.0f / sqrt(distSixth);
    const float sts = pjmass * invDistCube * TIMESTEP;
    *pivelx += xdistance * sts;
    *pively += ydistance * sts;
    *pivelz += zdistance * sts;
}

__global__ void aosoa_update_shared(particle_block *particles) {
    // TODO check for not multiple of 32
    __shared__ particle_block mainBlock;
    __shared__ particle_block otherBlock;
    const int mainLane = threadIdx.x;
    if (threadIdx.x == 0) {
        mainBlock = particles[blockIdx.x];
    }
    for (int otherBlockIndex = 0; otherBlockIndex < BLOCKS; ++otherBlockIndex) {
        otherBlock = particles[otherBlockIndex];
        SYNC_THREADS;
        for (int otherLane = 0; otherLane < LANES; ++otherLane) {
            pPInteraction(mainBlock.pos.x[mainLane],
                          mainBlock.pos.y[mainLane],
                          mainBlock.pos.z[mainLane],
                          &mainBlock.vel.x[mainLane],
                          &mainBlock.vel.y[mainLane],
                          &mainBlock.vel.z[mainLane],
                          otherBlock.pos.x[otherLane],
                          otherBlock.pos.y[otherLane],
                          otherBlock.pos.z[otherLane],
                          otherBlock.mass[otherLane]);
        }
    }
}

__global__ void aosoa_update(particle_block *particles) {
    // TODO check for not multiple of 32
    particle_block mainBlock = particles[blockIdx.x];
    particle_block otherBlock;
    const int mainLane = threadIdx.x;
    for (int otherBlockIndex = 0; otherBlockIndex < BLOCKS; ++otherBlockIndex) {
        otherBlock = particles[otherBlockIndex];
        for (int otherLane = 0; otherLane < LANES; ++otherLane) {
            pPInteraction(mainBlock.pos.x[mainLane],
                          mainBlock.pos.y[mainLane],
                          mainBlock.pos.z[mainLane],
                          &mainBlock.vel.x[mainLane],
                          &mainBlock.vel.y[mainLane],
                          &mainBlock.vel.z[mainLane],
                          otherBlock.pos.x[otherLane],
                          otherBlock.pos.y[otherLane],
                          otherBlock.pos.z[otherLane],
                          otherBlock.mass[otherLane]);
        }
    }
}

// Todo find a way to use more threads per block and less blocks
__global__ void aosoa_move(struct particle_block *particle_block) {
    int block_index = blockIdx.x;
    int i = threadIdx.x;

    struct particle_block block = particle_block[block_index];
    block.pos.x[i] += block.vel.x[i] * TIMESTEP;
    block.pos.y[i] += block.vel.y[i] * TIMESTEP;
    block.pos.z[i] += block.vel.z[i] * TIMESTEP;
}

__global__ void aosoa_randNormal(particle_block* particle_blocks) {
    int block = blockIdx.x;
    int lane = threadIdx.x;
    hiprandState state;
    hiprand_init(1337, lane, 0, &state);
  
    // iterate over pos-array inside the struct, then over the vel-arrays, then mass-array
    particle_blocks[block].pos.x[lane] = hiprand_normal(&state);;
    particle_blocks[block].pos.y[lane] = hiprand_normal(&state);;
    particle_blocks[block].pos.z[lane] = hiprand_normal(&state);;
    particle_blocks[block].vel.x[lane] = hiprand_normal(&state) / 10.0f;
    particle_blocks[block].vel.y[lane] = hiprand_normal(&state) / 10.0f;
    particle_blocks[block].vel.z[lane] = hiprand_normal(&state) / 10.0f;

    particle_blocks[block].mass[lane] = hiprand_normal(&state) / 100.0f;
}


void aosoa_run() {

    // "allocate" mem
    struct particle_block* particle_block_host = (particle_block*) malloc(BLOCKS* sizeof(particle_block));
    struct particle_block *particle_block_device;

    // TODO Corrrect omitted particels if PROBLEMSIZE not multiple of LANES
    // fill with random values
    // iterate over the structs 'stru' in the array
    /*
    for (int stru = 0; stru < BLOCKS; ++stru) {
        // iterate over pos-array inside the struct, then over the vel-arrays, then mass-array
        for (int l = 0; l < LANES; ++l) {
            particle_block_host[stru].pos.x[l] = (float) rand();
            particle_block_host[stru].pos.y[l] = (float) rand();
            particle_block_host[stru].pos.z[l] = (float) rand();

            particle_block_host[stru].vel.x[l] = (float) rand() / 10.0f;
            particle_block_host[stru].vel.y[l] = (float) rand() / 10.0f;
            particle_block_host[stru].vel.z[l] = (float) rand() / 10.0f;

            particle_block_host[stru].mass[l] = (float) rand() / 100.0f;
        }
    }*/

   


    int datasize = BLOCKS*sizeof(particle_block);
    HANDLE_ERROR(hipMalloc(&particle_block_device, datasize));
    // HANDLE_ERROR(hipMemcpy(particle_block_device, particle_block_host, datasize, hipMemcpyHostToDevice));
    aosoa_randNormal <<<BLOCKS, LANES >> > (particle_block_device);
    // init event management
    hipEvent_t start_update, stop_update;
    HANDLE_ERROR(hipEventCreate(&start_update));
    HANDLE_ERROR(hipEventCreate(&stop_update));

    hipEvent_t start_update_shared, stop_update_shared;
    HANDLE_ERROR(hipEventCreate(&start_update_shared));
    HANDLE_ERROR(hipEventCreate(&stop_update_shared));
    
    hipEvent_t start_move, stop_move;
    HANDLE_ERROR(hipEventCreate(&start_move));
    HANDLE_ERROR(hipEventCreate(&stop_move));
    float sum_move = 0, sum_update = 0, sum_update_shared = 0;
    float time_update, time_update_shared, time_move;
    for (int i = 0; i < STEPS; ++i) {
        // call update
        HANDLE_ERROR(hipEventRecord(start_update, 0));
        aosoa_update<<<(PROBLEMSIZE + LANES - 1) / LANES, LANES>>>(particle_block_device);
        HANDLE_ERROR(hipEventRecord(stop_update, 0));
        
        HANDLE_ERROR(hipEventRecord(start_update_shared, 0));
        aosoa_update_shared<<<(PROBLEMSIZE + LANES - 1) / LANES, LANES>>>(particle_block_device);
        HANDLE_ERROR(hipEventRecord(stop_update_shared, 0));

        // call move
        HANDLE_ERROR(hipEventRecord(start_move, 0));
        aosoa_move << <(PROBLEMSIZE + LANES - 1) / LANES, LANES >> > (particle_block_device);
        HANDLE_ERROR(hipEventRecord(stop_move));

        HANDLE_ERROR(hipEventSynchronize(stop_move));
        HANDLE_ERROR(hipEventElapsedTime(&time_update, start_update, stop_update));
        HANDLE_ERROR(hipEventElapsedTime(&time_update_shared, start_update_shared, stop_update_shared));
        HANDLE_ERROR(hipEventElapsedTime(&time_move, start_move, stop_move));
        printf("AoSoA\t%3.4fms\t%3.4fms\t%3.4fms\n", time_update,time_update_shared, time_move);
        sum_move += time_move;
        sum_update += time_update;
        sum_update_shared += time_update_shared;
    }
    printf("AVG:\t%3.4fms\t%3.4fms\t%3.6fms\n\n", sum_update / STEPS, sum_update_shared / STEPS,  sum_move / STEPS);

    // maybe write back

    // free mem
    free(particle_block_host);
    HANDLE_ERROR(hipFree(particle_block_device));
    HANDLE_ERROR(hipDeviceReset());
}
